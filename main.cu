/*
 * vim: ts=8:sw=8:tw=79:noet
 *
 * Copyright (c) 2013, Colin Patrick McCabe
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 * 
 * Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>

#include <stdio.h>

#define ARR_SIZE 5

#define EXPECT_CUDASUCCESS(x) \
	do { \
		hipError_t err = x; \
		if (x != hipSuccess) { \
			fprintf(stderr, "error %d (%s) on line %d of %s\n", \
				err, hipGetErrorString(err), \
				__LINE__, __FILE__); \
			exit(1); \
		} \
	} while (0);

static int *gd_a, *gd_b, *gd_c;

__global__ void add(int *a, int *b, int *c)
{
	int tid = blockIdx.x;
	if (tid < ARR_SIZE) {
		c[tid] = a[tid]  + b[tid];
	}
}

static void print_vector(const int * const a, size_t len)
{
	size_t i;
	const char *prefix = "";

	for (i = 0; i < len; i++) {
		printf("%s%d", prefix, a[i]);
		prefix = ", ";
	}
}

int main(void)
{
	int i, a[ARR_SIZE], b[ARR_SIZE], c[ARR_SIZE];
	for (i = 0; i < ARR_SIZE; i++) {
		a[i] = i;
	}
	for (i = 0; i < ARR_SIZE; i++) {
		b[i] = 1;
	}
	EXPECT_CUDASUCCESS(hipMalloc((void**)&gd_a,
			sizeof(int) * ARR_SIZE));
	EXPECT_CUDASUCCESS(hipMemcpy(gd_a, &a,
			sizeof(int) * ARR_SIZE, hipMemcpyHostToDevice));
	EXPECT_CUDASUCCESS(hipMalloc((void**)&gd_b,
			sizeof(int) * ARR_SIZE));
	EXPECT_CUDASUCCESS(hipMemcpy(gd_b, &b,
			sizeof(int) * ARR_SIZE, hipMemcpyHostToDevice));
	EXPECT_CUDASUCCESS(hipMalloc((void**)&gd_c,
			sizeof(int) * ARR_SIZE));
	add<<<ARR_SIZE, 1>>>(gd_a, gd_b, gd_c);
	EXPECT_CUDASUCCESS(hipMemcpy(c, gd_c,
			sizeof(int) * ARR_SIZE, hipMemcpyDeviceToHost));
	printf("initial vector a: ");
	print_vector(a, ARR_SIZE);
	printf("\ninitial vector b: ");
	print_vector(b, ARR_SIZE);
	printf("\nfinal vector c: ");
	print_vector(c, ARR_SIZE);
	printf("\n");

	hipFree(gd_a);
	hipFree(gd_b);
	hipFree(gd_c);
	return EXIT_SUCCESS;
}
